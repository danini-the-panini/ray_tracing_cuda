#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <hipcub/hipcub.hpp>

#define PI 3.14159265358979323846

#include "sphere_list.cuh"
#include "camera.cuh"

#define NX 1024
#define NY 512
#define NS 100
#define MAX_DEPTH 24

__device__ vec3 sky(const ray &r) {
    vec3 unit_direction = unit_vector(r.direction());
    float t = 0.5 * unit_direction.y() + 1.0;
    return (1.0-t) * vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
}

__device__ vec3 color(hiprandState &local_state, const ray &r, sphere_list *world) {
    hit_record rec;
    ray next_ray = r;
    ray scattered;
    vec3 attenuation[MAX_DEPTH];
    int num_hits = 0;
    vec3 col = vec3(0.0, 0.0, 0.0);

    for (int i = 0; i < MAX_DEPTH; i++) {
        if (hit(world, next_ray, 0.001, FLT_MAX, rec)) {
            if (rec.mat_ptr->scatter(local_state, next_ray, rec, attenuation[i], scattered)) {
                next_ray = scattered;
                num_hits++;
            } else {
                break;
            }
        } else {
            col = sky(scattered);
            break;
        }
    }

    for (int i = num_hits-1; i >= 0; i--) {
        col *= attenuation[i];
    }

    return col;
}

__global__ void setup_kernel(hiprandState * state, unsigned long seed)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init((seed<<20)+id, 0, 0, &state[id]);
}

__device__ vec3 add_vec3(const vec3 &v1, const vec3 &v2) {
    return v1 + v2;
}

__global__ void kernel(hiprandState* global_state, int nx, int ny, sphere_list *world, const camera &cam, unsigned char *out) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState local_state = global_state[id];

#if NS > 1
    typedef hipcub::BlockReduce<vec3, NS> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
#endif

    vec3 lower_left_corner = vec3(-2.0, -1.0, -1.0);
    vec3 horizontal = vec3(4.0, 0.0, 0.0);
    vec3 vertical = vec3(0.0, 2.0, 0.0);
    vec3 origin = vec3(0.0, 0.0, 0.0);

    int n = blockIdx.x;
    int j = ny-n/nx-1;
    int i = n%nx;

    float u = (float(i) + hiprand_uniform(&local_state)) / float(nx);
    float v = (float(j) + hiprand_uniform(&local_state)) / float(ny);

    ray r = cam.get_ray(local_state, u, v);
    vec3 col = color(local_state, r, world);
    col/=float(NS);

#if NS > 1
    col = BlockReduce(temp_storage).Reduce(col, add_vec3);
#endif

    col = vec3(sqrt(col.r()), sqrt(col.g()), sqrt(col.b()));
    if (threadIdx.x == 0) {
        out[n*3+0] = int(255.99*col.r());
        out[n*3+1] = int(255.99*col.g());
        out[n*3+2] = int(255.99*col.b());
    }
}

int main(void) {
    hiprandState* device_states;
    hipMalloc(&device_states, NX*NY*NS*sizeof(hiprandState));

    setup_kernel<<<NX*NY,NS>>>(device_states, time(NULL));

    size_t BUFFER_SIZE = sizeof(unsigned char)*NX*NY*3;

    printf("P3\n%d %d\n255\n", NX, NY);

    sphere_list *world = make_shared_sphere_list(5);
    sphere **list = world->list;
    list[0] = new sphere(vec3(0,0,-1), 0.5, new material(LAMBERTIAN, vec3(0.1, 0.2, 0.5)));
    list[1] = new sphere(vec3(0,-100.5,-1), 100, new material(LAMBERTIAN, vec3(0.8, 0.6, 0.2)));
    list[2] = new sphere(vec3(1,0,-1), 0.5, new material(METAL, vec3(0.8, 0.6, 0.2), 0.1));
    list[3] = new sphere(vec3(-1,0,-1), 0.5, new material(DIELECTRIC, 1.5));
    list[4] = new sphere(vec3(-1,0,-1), -0.45, new material(DIELECTRIC, 1.5));
    
    unsigned char *out = (unsigned char*)malloc(BUFFER_SIZE); // host ouput
    unsigned char *d_out; // device output
    hipMalloc(&d_out, BUFFER_SIZE);

    vec3 lookfrom(3,3,2);
    vec3 lookat(0,0,-1);
    float dist_to_focus = (lookfrom-lookat).length();
    float aperture = 1.0;

    camera *cam = new camera(lookfrom, lookat, vec3(0,1,0), 20, float(NX)/float(NY), aperture, dist_to_focus);

    kernel<<<NX*NY,NS>>>(device_states, NX, NY, world, *cam, d_out);

    hipMemcpy(out, d_out, BUFFER_SIZE, hipMemcpyDeviceToHost);

    for (int n = 0; n < NX*NY; n++) {
        printf("%d %d %d\n", out[n*3+0], out[n*3+1], out[n*3+2]);
    }

    hipFree(d_out);
    free(out);

    clean_up_sphere_list(world);
    delete cam;

    return 0;
}